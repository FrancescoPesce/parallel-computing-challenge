#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

int main(int argc, char const *argv[])
{
    // retrieve some info about the CUDA device
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      std::cout << "Device Number: " << i << std::endl;
      std::cout << "  Device name: " << prop.name << std::endl;
      std::cout << "  max Blocks Per MultiProcessor: " << prop.maxBlocksPerMultiProcessor << std::endl;
      std::cout << "  max Threads Per MultiProcessor: " << prop.maxThreadsPerMultiProcessor << std::endl;
      std::cout << "  max Threads Per Block: " << prop.maxThreadsPerBlock << std::endl;
      std::cout << "  num SM: " << prop.multiProcessorCount << std::endl;
      std::cout << "  num bytes sharedMem Per Block: " << prop.sharedMemPerBlock << std::endl;
      std::cout << "  num bytes sharedMem Per Multiprocessor: " << prop.sharedMemPerMultiprocessor << std::endl;
      std::cout << "  Memory Clock Rate (KHz): " << prop.memoryClockRate << std::endl;
      std::cout << "  Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
      std::cout << "  Peak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << std::endl << std::endl;
    }
    
    return 0;
}
